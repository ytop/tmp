#include "hip/hip_runtime.h"
/**
 * Copyright 2020 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "runtime/device/gpu/cuda_common.h"
#include "include/hip/hip_fp16.h"

#include "backend/kernel_compiler/gpu/cuda_impl/topk_lib.cuh"
#include "backend/kernel_compiler/gpu/cuda_impl/argmaxwithvalue_impl.cuh"
#include <limits>
#include <algorithm>

template <typename T, typename S, int threads_per_block>
__global__ void L2ArgMax(int outer_size, int inner_size, int stride, const T *input, T *output, S *output_index) {
  constexpr int kNumWarps = threads_per_block / kWarpSize;
  const int threads_per_warp = 32;
  const T init_K = static_cast<T>(-9999);
  constexpr S init_V = static_cast<S>(-1);

  for (int t_idx = blockIdx.x * blockDim.x + threadIdx.x; t_idx < blockDim.x * outer_size * stride;
       t_idx += blockDim.x * gridDim.x) {
    __shared__ T shared_K[kNumWarps];
    __shared__ S shared_V[kNumWarps];

    T warp_K_top = init_K;
    int outer_id = t_idx / blockDim.x / stride;
    int inner_id = t_idx / blockDim.x % stride;

    T threadK = init_K;
    S threadV = init_V;

    int laneId = GetLaneId();
    int warpId = threadIdx.x / kWarpSize;  // 0,1,2 or 3

    // sync till all threads init done
    __syncwarp();

    int i = threadIdx.x;
    for (; i < inner_size; i += threads_per_block) {
      auto &k = input[outer_id * inner_size * stride + i * stride + inner_id];
      auto &v = i;
      if (Cmp<T>::gt(k, warp_K_top)) {
        {
          threadK = k;
          threadV = v;
          warp_K_top = k;
        }
      }
    }
    __syncwarp();

    for (int offset = threads_per_warp / 2; offset > 0; offset /= 2) {
      T other_K = __shfl_down_sync(0xffffffff, threadK, offset);
      S other_V = __shfl_down_sync(0xffffffff, threadV, offset);

      bool small_compare_descend = Cmp<T>::lt(threadK, other_K);
      ConditionAssign(small_compare_descend, &threadK, other_K);
      ConditionAssign(small_compare_descend, &threadV, other_V);
    }
    shared_K[warpId] = threadK;
    shared_V[warpId] = threadV;
    __syncthreads();

    _Pragma("unroll") for (int offset = kNumWarps / 2; offset > 0; offset /= 2) {
      int pos = threadIdx.x;
      if ( pos >= (kNumWarps/offset)) break; 
      L2CompareAndSwap<T, S, false>(shared_K, shared_V, pos, pos + offset);
    }
    __syncwarp();

    output[outer_id * stride + inner_id] = shared_K[0];
    output_index[outer_id * stride + inner_id] = shared_V[0];
  }
}

template <typename T, typename S>
void ArgmaxWithValue(int outer_size, int inner_size, int stride, const T *input, T *output, S *output_index,
                     hipStream_t stream) {
  int block_num_limit = outer_size * stride < 1024 ? outer_size * stride : 1024;
  L2ArgMax<T, S, 256><<<GET_BLOCKS((256 * block_num_limit)), 256, 0, stream>>>(outer_size, inner_size, stride, input,
                                                                               output, output_index);
}

template <typename T, typename S>
void CalArgmaxWithValue(const T *input, const int bound, const int outerSize, const int innerSize, S *index, T *output,
                        hipStream_t hip_stream) {
  ArgmaxWithValue(outerSize, bound, innerSize, input, output, index, hip_stream);
  return;
}

template void CalArgmaxWithValue<float, int>(const float *input, const int bound_, const int outerSize_,
                                             const int innerSize_, int *index, float *output, hipStream_t hip_stream);
template void CalArgmaxWithValue<half, int>(const half *input, const int bound_, const int outerSize_,
                                            const int innerSize_, int *index, half *output, hipStream_t hip_stream);
